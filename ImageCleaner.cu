#include "hip/hip_runtime.h"
#include "ImageCleaner.h"

#ifndef SIZEX
#error Please define SIZEX.
#endif
#ifndef SIZEY
#error Please define SIZEY.
#endif

#define SIZE SIZEX
#define PI     3.14159256f
#define TWO_PI 6.28318530f

//----------------------------------------------------------------
// TODO:  CREATE NEW KERNELS HERE.  YOU CAN PLACE YOUR CALLS TO
//        THEM IN THE INDICATED SECTION INSIDE THE 'filterImage'
//        FUNCTION.
//
// BEGIN ADD KERNEL DEFINITIONS
//----------------------------------------------------------------


// __device__ char forwardFFT_arbitrary_radix(int radix, float (*real)[SIZE], float (*imag)[SIZE])
// {
//   int span = SIZE / radix;

//   for (int unit_size = 1; unit_size < SIZE ; unit_size *= span)
//   {
//     int pos_in_unit = threadIdx.x % unit_size;
//   }

// }

__device__ char forwardFFT_radix4(float (*real)[SIZE], float (*imag)[SIZE])
{

  __shared__ float roots_real_local[SIZE];
  __shared__ float roots_imag_local[SIZE];


  float angle = - TWO_PI * threadIdx.x / SIZE;
  roots_real_local[threadIdx.x] = __cosf(angle);
  roots_imag_local[threadIdx.x] = __sinf(angle);
  

  __syncthreads();

  char curr = 0;
  char next = 1;

  // int span = SIZE >> 2;
  // int temp;

  for (int unit_size = 1; unit_size < SIZE ; unit_size <<= 2)
  {
    int pos_in_unit = threadIdx.x % unit_size;
    // temp = pos_in_unit * (SIZE >> 2) / unit_size; // twiddle index
    // float twiddle1k_real = roots_real_local[temp];
    // float twiddle1k_imag = roots_imag_local[temp];
    // float twiddle2k_real = roots_real_local[temp << 1];
    // float twiddle2k_imag = roots_imag_local[temp << 1];
    // float twiddle3k_real = roots_real_local[3*temp];
    // float twiddle3k_imag = roots_imag_local[3*temp];


    if (threadIdx.x < (SIZE >> 2))
    {
      //x1 = x1 + twiddle1k * x2 + twiddle2k * x3 + twiddle3k * x4

      //no need to store x1, already there
      __syncthreads();

      int ind1 = threadIdx.x;
      int ind2 = threadIdx.x + (SIZE >> 2);
      int ind3 = threadIdx.x + (SIZE >> 1);
      int ind4 = threadIdx.x + (SIZE >> 1) + (SIZE >> 2); 

      int new_pos = ((threadIdx.x - pos_in_unit) << 2) + pos_in_unit; //new index of x1
      real[next][new_pos] = real[curr][ind1] + real[curr][ind2] + real[curr][ind3] + real[curr][ind4];
      imag[next][new_pos] = imag[curr][ind1] + imag[curr][ind2] + imag[curr][ind3] + imag[curr][ind4];

      // temp = threadIdx.x + (SIZE >> 2); // index of x2
      // int ind3 = threadIdx.x + (SIZE >> 1);
      // int ind4 = ind3 + (SIZE >> 2); 
      // float r2 = real[curr][temp];
      // float r3 = real[curr][ind3];
      // float r4 = real[curr][ind4];
      // float i2 = imag[curr][temp];
      // float i3 = imag[curr][ind3];
      // float i4 = imag[curr][ind4];
      // temp = ((threadIdx.x - pos_in_unit) << 2) + pos_in_unit; //new index of x1
      // real[next][temp] = real[curr][threadIdx.x] + (twiddle1k_real * r2 - twiddle1k_imag * i2) + (twiddle2k_real * r3 - twiddle2k_imag * i3) + (twiddle3k_real * r4 - twiddle3k_imag * i4);
      // imag[next][temp] = imag[curr][threadIdx.x] + (twiddle1k_real * i2 + twiddle1k_imag * r2) + (twiddle2k_real * i3 + twiddle2k_imag * r3) + (twiddle3k_real * i4 + twiddle3k_imag * r4);
    }
    else if (threadIdx.x < SIZE >> 1)
    {
      //x2 = x1 - j*twiddle1k * x2 - twiddle2k * x3 + j twiddle3k * x4
      int twiddle_index = pos_in_unit * (SIZE >> 2) / unit_size;
      float twiddle1k_real = roots_real_local[twiddle_index];
      float twiddle1k_imag = roots_imag_local[twiddle_index];

      //store twiddle1k * x2
      float r = real[curr][threadIdx.x], i = imag[curr][threadIdx.x];
      real[curr][threadIdx.x] = twiddle1k_real * r - twiddle1k_imag * i;
      imag[curr][threadIdx.x] = twiddle1k_imag * r + twiddle1k_real * i;
      __syncthreads();

      int ind1 = threadIdx.x - (SIZE >> 2);
      int ind2 = threadIdx.x;
      int ind3 = threadIdx.x + (SIZE >> 2);
      int ind4 = threadIdx.x + (SIZE >> 1); 

      int new_pos = ((threadIdx.x - pos_in_unit - (SIZE >> 2)) << 2) + (unit_size + pos_in_unit) ; //new index of x2
      real[next][new_pos] = real[curr][ind1] + imag[curr][ind2] - real[curr][ind3] - imag[curr][ind4];
      imag[next][new_pos] = imag[curr][ind1] - real[curr][ind2] - imag[curr][ind3] + real[curr][ind4];


      // temp = threadIdx.x - (SIZE >> 2); // index of x1
      // int ind3 = threadIdx.x + (SIZE >> 2);
      // int ind4 = threadIdx.x + (SIZE >> 1); 
      // float r1 = real[curr][temp];
      // // float r2 = real[curr][threadIdx.x];
      // float r3 = real[curr][ind3];
      // float r4 = real[curr][ind4];
      // float i1 = imag[curr][temp];
      // // float i2 = imag[curr][threadIdx.x];
      // float i3 = imag[curr][ind3];
      // float i4 = imag[curr][ind4];
      // temp = ((threadIdx.x - pos_in_unit - (SIZE >> 2)) << 2) + (unit_size + pos_in_unit) ; //new index of x2
      // real[next][temp] = r1 + (twiddle1k_real * imag[curr][threadIdx.x] + twiddle1k_imag * real[curr][threadIdx.x]) - (twiddle2k_real * r3 - twiddle2k_imag * i3) - (twiddle3k_real * i4 + twiddle3k_imag * r4);
      // imag[next][temp] = i1 - (twiddle1k_real * real[curr][threadIdx.x] - twiddle1k_imag * imag[curr][threadIdx.x]) - (twiddle2k_real * i3 + twiddle2k_imag * r3) + (twiddle3k_real * r4 - twiddle3k_imag * i4);

    }
    else if (threadIdx.x < (SIZE >> 1) + (SIZE >> 2))
    {
      //x3 = x1 - twiddle1k * x2 + twiddle2k * x3 - twiddle3k * x4
      int twiddle_index = pos_in_unit * (SIZE >> 1) / unit_size;
      float twiddle2k_real = roots_real_local[twiddle_index];
      float twiddle2k_imag = roots_imag_local[twiddle_index];

      //store twiddle2k * x3
      float r = real[curr][threadIdx.x], i = imag[curr][threadIdx.x];
      real[curr][threadIdx.x] = twiddle2k_real * r - twiddle2k_imag * i;
      imag[curr][threadIdx.x] = twiddle2k_imag * r + twiddle2k_real * i;
      __syncthreads();

      int ind1 = threadIdx.x - (SIZE >> 1);
      int ind2 = threadIdx.x - (SIZE >> 2);
      int ind3 = threadIdx.x;
      int ind4 = threadIdx.x + (SIZE >> 2); 

      int new_pos = ((threadIdx.x - pos_in_unit - (SIZE >> 1)) << 2) + ((unit_size >> 1) + pos_in_unit) ; //new index of x3
      real[next][new_pos] = real[curr][ind1] - real[curr][ind2] + real[curr][ind3] - real[curr][ind4];
      imag[next][new_pos] = imag[curr][ind1] - imag[curr][ind2] + imag[curr][ind3] - imag[curr][ind4];

      // temp = threadIdx.x - (SIZE >> 1); // index of x1
      // int ind2 = threadIdx.x - (SIZE >> 2);
      // int ind4 = threadIdx.x + (SIZE >> 2); 
      // float r1 = real[curr][temp];
      // float r2 = real[curr][ind2];
      // // float r3 = real[curr][threadIdx.x];
      // float r4 = real[curr][ind4];
      // float i1 = imag[curr][temp];
      // float i2 = imag[curr][ind2];
      // // float i3 = imag[curr][threadIdx.x];
      // float i4 = imag[curr][ind4];
      // temp = ((threadIdx.x - pos_in_unit - (SIZE >> 1)) << 2) + ((unit_size << 1) + pos_in_unit) ; //new index of x3
      // real[next][temp] = r1 - (twiddle1k_real * r2 - twiddle1k_imag * i2) + (twiddle2k_real * real[curr][threadIdx.x] - twiddle2k_imag * imag[curr][threadIdx.x]) - (twiddle3k_real * r4 - twiddle3k_imag * i4);
      // imag[next][temp] = i1 - (twiddle1k_real * i2 + twiddle1k_imag * r2) + (twiddle2k_real * imag[curr][threadIdx.x] + twiddle2k_imag * real[curr][threadIdx.x]) - (twiddle3k_real * i4 + twiddle3k_imag * r4);
    }
    else
    {
      //x4 = x1 +j twiddle1k * x2 - twiddle2k * x3 -j twiddle3k * x4
      int twiddle_index = pos_in_unit * 3 * (SIZE >> 2) / unit_size;
      float twiddle3k_real = roots_real_local[twiddle_index];
      float twiddle3k_imag = roots_imag_local[twiddle_index];

      //store twiddle1k * x2
      float r = real[curr][threadIdx.x], i = imag[curr][threadIdx.x];
      real[curr][threadIdx.x] = twiddle3k_real * r - twiddle3k_imag * i;
      imag[curr][threadIdx.x] = twiddle3k_imag * r + twiddle3k_real * i;
      __syncthreads();

      int ind1 = threadIdx.x - (SIZE >> 2) - (SIZE >> 1);
      int ind2 = threadIdx.x - (SIZE >> 1);
      int ind3 = threadIdx.x - (SIZE >> 2);
      int ind4 = threadIdx.x; 

      int new_pos = ((threadIdx.x - pos_in_unit - 3 * (SIZE >> 2)) << 2) + (3 * unit_size + pos_in_unit) ; //new index of x4
      real[next][new_pos] = real[curr][ind1] - imag[curr][ind2] - real[curr][ind3] + imag[curr][ind4];
      imag[next][new_pos] = imag[curr][ind1] + real[curr][ind2] - imag[curr][ind3] - real[curr][ind4];

      // temp = threadIdx.x - 3 * (SIZE >> 2); // index of x1
      // int ind2 = threadIdx.x - (SIZE >> 1);
      // int ind3 = threadIdx.x - (SIZE >> 2); 
      // float r1 = real[curr][temp];
      // float r2 = real[curr][ind2];
      // float r3 = real[curr][ind3];
      // // float r4 = real[curr][threadIdx.x];
      // float i1 = imag[curr][temp];
      // float i2 = imag[curr][ind2];
      // float i3 = imag[curr][ind3];
      // // float i4 = imag[curr][threadIdx.x];
      // temp = ((threadIdx.x - pos_in_unit - 3 * (SIZE >> 2)) << 2) + (3 * unit_size + pos_in_unit) ; //new index of x4
      // real[next][temp] = r1 - (twiddle1k_real * i2 + twiddle1k_imag * r2) - (twiddle2k_real * r3 - twiddle2k_imag * i3) + (twiddle3k_real * imag[curr][threadIdx.x] + twiddle3k_imag * real[curr][threadIdx.x]);
      // imag[next][temp] = i1 + (twiddle1k_real * r2 - twiddle1k_imag * i2) - (twiddle2k_real * i3 + twiddle2k_imag * r3) - (twiddle3k_real * real[curr][threadIdx.x] - twiddle3k_imag * imag[curr][threadIdx.x]);

    }
  __syncthreads();
    next = curr;
    curr = 1 - curr;
  }
  return curr;
}

__device__ char inverseFFT_radix4(float (*real)[SIZE], float (*imag)[SIZE])
{

  __shared__ float roots_real_local[SIZE];
  __shared__ float roots_imag_local[SIZE];


  float angle = TWO_PI * threadIdx.x / SIZE;
  roots_real_local[threadIdx.x] = __cosf(angle);
  roots_imag_local[threadIdx.x] = __sinf(angle);
  

  __syncthreads();

  char curr = 0;
  char next = 1;

  // int span = SIZE >> 2;
  int temp;

  for (int unit_size = 1; unit_size < SIZE ; unit_size <<= 2)
  {
    int pos_in_unit = threadIdx.x % unit_size;
    temp = pos_in_unit * (SIZE >> 2) / unit_size; // twiddle index
    float twiddle1k_real = roots_real_local[temp];
    float twiddle1k_imag = roots_imag_local[temp];
    float twiddle2k_real = roots_real_local[temp << 1];
    float twiddle2k_imag = roots_imag_local[temp << 1];
    float twiddle3k_real = roots_real_local[3*temp];
    float twiddle3k_imag = roots_imag_local[3*temp];


    if (threadIdx.x < (SIZE >> 2))
    {
      //x1 = x1 + twiddle1k * x2 + twiddle2k * x3 + twiddle3k * x4
      temp = threadIdx.x + (SIZE >> 2); // index of x2
      int ind3 = threadIdx.x + (SIZE >> 1);
      int ind4 = ind3 + (SIZE >> 2); 
      float r2 = real[curr][temp];
      float r3 = real[curr][ind3];
      float r4 = real[curr][ind4];
      float i2 = imag[curr][temp];
      float i3 = imag[curr][ind3];
      float i4 = imag[curr][ind4];
      temp = ((threadIdx.x - pos_in_unit) << 2) + pos_in_unit; //new index of x1
      real[next][temp] = real[curr][threadIdx.x] + (twiddle1k_real * r2 - twiddle1k_imag * i2) + (twiddle2k_real * r3 - twiddle2k_imag * i3) + (twiddle3k_real * r4 - twiddle3k_imag * i4);
      imag[next][temp] = imag[curr][threadIdx.x] + (twiddle1k_real * i2 + twiddle1k_imag * r2) + (twiddle2k_real * i3 + twiddle2k_imag * r3) + (twiddle3k_real * i4 + twiddle3k_imag * r4);
    }
    else if (threadIdx.x < SIZE >> 1)
    {
      //x2 = x1 - j*twiddle1k * x2 - twiddle2k * x3 + j twiddle3k * x4
      temp = threadIdx.x - (SIZE >> 2); // index of x1
      int ind3 = threadIdx.x + (SIZE >> 2);
      int ind4 = threadIdx.x + (SIZE >> 1); 
      float r2 = real[curr][threadIdx.x];
      float r3 = real[curr][ind3];
      float r4 = real[curr][ind4];
      float i2 = imag[curr][threadIdx.x];
      float i3 = imag[curr][ind3];
      float i4 = imag[curr][ind4];
      temp = ((threadIdx.x - pos_in_unit - (SIZE >> 2)) << 2) + (unit_size + pos_in_unit) ; //new index of x2
      real[next][temp] = real[curr][temp] + (twiddle1k_real * i2 + twiddle1k_imag * r2) - (twiddle2k_real * r3 - twiddle2k_imag * i3) - (twiddle3k_real * i4 + twiddle3k_imag * r4);
      imag[next][temp] = imag[curr][temp] - (twiddle1k_real * r2 - twiddle1k_imag * i2) - (twiddle2k_real * i3 + twiddle2k_imag * r3) + (twiddle3k_real * r4 - twiddle3k_imag * i4);

    }
    else if (threadIdx.x < (SIZE >> 1) + (SIZE >> 2))
    {
      //x3 = x1 - twiddle1k * x2 + twiddle2k * x3 - twiddle3k * x4
      temp = threadIdx.x - (SIZE >> 1); // index of x1
      int ind2 = threadIdx.x - (SIZE >> 2);
      int ind4 = threadIdx.x + (SIZE >> 2); 
      float r2 = real[curr][ind2];
      float r3 = real[curr][threadIdx.x];
      float r4 = real[curr][ind4];
      float i2 = imag[curr][ind2];
      float i3 = imag[curr][threadIdx.x];
      float i4 = imag[curr][ind4];
      temp = ((threadIdx.x - pos_in_unit - (SIZE >> 1)) << 2) + ((unit_size >> 1) + pos_in_unit) ; //new index of x3
      real[next][temp] = real[curr][temp] - (twiddle1k_real * r2 - twiddle1k_imag * i2) + (twiddle2k_real * r3 - twiddle2k_imag * i3) - (twiddle3k_real * r4 - twiddle3k_imag * i4);
      imag[next][temp] = imag[curr][temp] - (twiddle1k_real * i2 + twiddle1k_imag * r2) + (twiddle2k_real * i3 + twiddle2k_imag * r3) - (twiddle3k_real * i4 + twiddle3k_imag * r4);
    }
    else
    {
      //x4 = x1 +j twiddle1k * x2 - twiddle2k * x3 -j twiddle3k * x4
      temp = threadIdx.x - 3 * (SIZE >> 2); // index of x1
      int ind2 = threadIdx.x - (SIZE >> 1);
      int ind3 = threadIdx.x - (SIZE >> 2); 
      float r2 = real[curr][ind2];
      float r3 = real[curr][ind3];
      float r4 = real[curr][threadIdx.x];
      float i2 = imag[curr][ind2];
      float i3 = imag[curr][ind3];
      float i4 = imag[curr][threadIdx.x];
      temp = ((threadIdx.x - pos_in_unit - 3 * (SIZE >> 2)) << 2) + (3 * unit_size + pos_in_unit) ; //new index of x3
      real[next][temp] = real[curr][temp] - (twiddle1k_real * i2 + twiddle1k_imag * r2) - (twiddle2k_real * r3 - twiddle2k_imag * i3) + (twiddle3k_real * i4 + twiddle3k_imag * r4);
      imag[next][temp] = imag[curr][temp] + (twiddle1k_real * r2 - twiddle1k_imag * i2) - (twiddle2k_real * i3 + twiddle2k_imag * r3) - (twiddle3k_real * r4 - twiddle3k_imag * i4);

    }
  __syncthreads();
    next = curr;
    curr = 1 - curr;
  }
  return curr;
}

__device__ char forwardFFT(int pos, float (*real)[SIZE], float (*imag)[SIZE])
{
  __shared__ float roots_real_local[SIZE/2];
  __shared__ float roots_imag_local[SIZE/2];

  if (pos < SIZE / 2)
  {
    float angle = - TWO_PI * pos / SIZE;
    roots_real_local[pos] = __cosf(angle);
    roots_imag_local[pos] = __sinf(angle);
  }


  __syncthreads();

  char curr = 0;
  char next = 1;

  int span = SIZE >> 1;
  int temp;

  for (int unit_size = 1; unit_size < SIZE ; unit_size <<= 1)
  {
    int pos_in_unit = pos % unit_size;
    temp = pos_in_unit * (SIZE >> 1) / unit_size; // twiddle index
    float twiddle_real = roots_real_local[temp];
    float twiddle_imag = roots_imag_local[temp];

    if (pos < span)
    {
      //x1 = x1 + twiddle * x2
      temp = pos + span; // index of x2
      float r1 = real[curr][pos];
      float r2 = real[curr][temp];
      float i1 = imag[curr][pos];
      float i2 = imag[curr][temp];
      temp = (pos << 1) - pos_in_unit; //new index of x1
      real[next][temp] = r1 + (twiddle_real * r2 - twiddle_imag * i2);
      imag[next][temp] = i1 + (twiddle_real * i2 + twiddle_imag * r2);
    }
    else
    {
      // x2 = x1 - twiddle *x2
      temp = pos - span; // index of x1
      float r1 = real[curr][temp];
      float r2 = real[curr][pos];
      float i1 = imag[curr][temp];
      float i2 = imag[curr][pos];
      temp = ((pos - span) << 1) - pos_in_unit + unit_size; //new index of x2
      real[next][temp] = r1 - (twiddle_real * r2 - twiddle_imag * i2);
      imag[next][temp] = i1 - (twiddle_real * i2 + twiddle_imag * r2);
    }
    __syncthreads();
    next = curr;
    curr = 1 - curr;
  }
  return curr;
}

__device__ char inverseFFT(int pos, float (*real)[SIZE], float (*imag)[SIZE])
{
  __shared__ float roots_real_local[SIZE/2];
  __shared__ float roots_imag_local[SIZE/2];

  if (pos < SIZE / 2)
  {
    float angle = TWO_PI * pos / SIZE;
    roots_real_local[pos] = __cosf(angle);
    roots_imag_local[pos] = __sinf(angle);
  }


  __syncthreads();

  char curr = 0;
  char next = 1;

  int span = SIZE >> 1;
  int temp;

  for (int unit_size = 1; unit_size < SIZE ; unit_size <<= 1)
  {
    int pos_in_unit = pos % unit_size;
    temp = pos_in_unit * (SIZE >> 1) / unit_size; // twiddle index
    float twiddle_real = roots_real_local[temp];
    float twiddle_imag = roots_imag_local[temp];

    if (pos < span)
    {
      //x1 = x1 + twiddle * x2
      temp = pos + span; // index of x2
      float r1 = real[curr][pos];
      float r2 = real[curr][temp];
      float i1 = imag[curr][pos];
      float i2 = imag[curr][temp];
      temp = (pos << 1) - pos_in_unit; //new index of x1
      real[next][temp] = r1 + (twiddle_real * r2 - twiddle_imag * i2);
      imag[next][temp] = i1 + (twiddle_real * i2 + twiddle_imag * r2);
    }
    else
    {
      // x2 = x1 - twiddle *x2
      temp = pos - span; // index of x1
      float r1 = real[curr][temp];
      float r2 = real[curr][pos];
      float i1 = imag[curr][temp];
      float i2 = imag[curr][pos];
      temp = ((pos - span) << 1) - pos_in_unit + unit_size; //new index of x2
      real[next][temp] = r1 - (twiddle_real * r2 - twiddle_imag * i2);
      imag[next][temp] = i1 - (twiddle_real * i2 + twiddle_imag * r2);
    }
    __syncthreads();
    next = curr;
    curr = 1 - curr;
  }
  return curr;
}

__global__ void forwardFFTRow(float *real_image, float *imag_image)
{
  int row = blockIdx.x;
  int col = threadIdx.x;

  __shared__ float real[2][SIZE];
  __shared__ float imag[2][SIZE];


  int offset = row * SIZE + col;

  real[0][col] = real_image[offset];
  imag[0][col] = imag_image[offset];


  char curr = forwardFFT_radix4(real, imag);

  real_image[offset] = real[curr][col];
  imag_image[offset] = imag[curr][col];
}

__global__ void inverseFFTRow(float *real_image, float *imag_image)
{
  int row = blockIdx.x;
  int col = threadIdx.x;

  __shared__ float real[2][SIZE];
  __shared__ float imag[2][SIZE];


  int offset = row * SIZE + col;

  real[0][col] = real_image[offset];
  imag[0][col] = imag_image[offset];


  char curr = inverseFFT(col, real, imag);

  real_image[offset] = real[curr][col] / SIZE;
  imag_image[offset] = imag[curr][col] / SIZE;
}

__global__ void forwardFFTCol(float *real_image, float *imag_image)
{
  int col = blockIdx.x;
  int row = threadIdx.x;
  if (col >= SIZE / 8)
  {
    col += 3 * SIZE / 4;
  }
  __shared__ float real[2][SIZE];
  __shared__ float imag[2][SIZE];

  real[0][row] = real_image[row * SIZE + col];
  imag[0][row] = imag_image[row * SIZE + col];

  char curr = forwardFFT(row, real, imag);

  real_image[row * SIZE + col] = real[curr][row];
  imag_image[row * SIZE + col] = imag[curr][row];
}

__global__ void inverseFFTCol(float *real_image, float *imag_image)
{
  int col = blockIdx.x;
  int row = threadIdx.x;
  if (col >= SIZE / 8)
  {
    col += 3 * SIZE / 4;
  }
  __shared__ float real[2][SIZE];
  __shared__ float imag[2][SIZE];

  real[0][row] = real_image[row * SIZE + col];
  imag[0][row] = imag_image[row * SIZE + col];

  char curr = inverseFFT(row, real, imag);

  real_image[row * SIZE + col] = real[curr][row] / SIZE;
  imag_image[row * SIZE + col] = imag[curr][row] / SIZE;
}

__global__ void filter(float *real_image, float *imag_image)
{
  int row = blockIdx.x;
  int col = threadIdx.x;

  int eighth = SIZE / 8;
  int seven_eighth = SIZE - eighth;

  if ((row >= eighth && row < seven_eighth) || (col >= eighth && col < seven_eighth))
  {
    real_image[row * SIZE + col] = 0.f;
    imag_image[row * SIZE + col] = 0.f;
  }
}


//----------------------------------------------------------------
// END ADD KERNEL DEFINTIONS
//----------------------------------------------------------------

__host__ float filterImage(float *real_image, float *imag_image, int size_x, int size_y)
{
  // check that the sizes match up
  assert(size_x == SIZEX);
  assert(size_y == SIZEY);

  int matSize = size_x * size_y * sizeof(float);

  // These variables are for timing purposes
  float transferDown = 0, transferUp = 0, execution = 0;
  hipEvent_t start,stop;

  float fftr = 0.f, fftc = 0.f, ifftr = 0.f, ifftc = 0.f, filter_time = 0.f, stream_creation = 0.f;
  hipEvent_t start_bis, stop_bis;

  CUDA_ERROR_CHECK(hipEventCreate(&start));
  CUDA_ERROR_CHECK(hipEventCreate(&stop));

  CUDA_ERROR_CHECK(hipEventCreate(&start_bis));
  CUDA_ERROR_CHECK(hipEventCreate(&stop_bis));

  // Create a stream and initialize it
  hipStream_t filterStream;
  CUDA_ERROR_CHECK(hipStreamCreate(&filterStream));

  // Alloc space on the device
  float *device_real, *device_imag;
  CUDA_ERROR_CHECK(hipMalloc((void**)&device_real, matSize));
  CUDA_ERROR_CHECK(hipMalloc((void**)&device_imag, matSize));

  // Start timing for transfer down
  CUDA_ERROR_CHECK(hipEventRecord(start,filterStream));
  
  // Here is where we copy matrices down to the device 
  // CUDA_ERROR_CHECK(hipMemcpy(device_real,real_image,matSize,hipMemcpyHostToDevice));
  // CUDA_ERROR_CHECK(hipMemcpy(device_imag,imag_image,matSize,hipMemcpyHostToDevice));
  
  // Stop timing for transfer down
  CUDA_ERROR_CHECK(hipEventRecord(stop,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&transferDown,start,stop));

  // Start timing for the execution
  CUDA_ERROR_CHECK(hipEventRecord(start,filterStream));

  //----------------------------------------------------------------
  // TODO: YOU SHOULD PLACE ALL YOUR KERNEL EXECUTIONS
  //        HERE BETWEEN THE CALLS FOR STARTING AND
  //        FINISHING TIMING FOR THE EXECUTION PHASE
  // BEGIN ADD KERNEL CALLS
  //----------------------------------------------------------------

  // This is an example kernel call, you should feel free to create
  // as many kernel calls as you feel are needed for your program
  // Each of the parameters are as follows:
  //    1. Number of thread blocks, can be either int or dim3 (see CUDA manual)
  //    2. Number of threads per thread block, can be either int or dim3 (see CUDA manual)
  //    3. Always should be '0' unless you read the CUDA manual and learn about dynamically allocating shared memory
  //    4. Stream to execute kernel on, should always be 'filterStream'
  //
  // Also note that you pass the pointers to the device memory to the kernel call

  // float *pinned_real_image, *pinned_imag_image;
  // CUDA_ERROR_CHECK(hipHostMalloc((void **) &pinned_real_image, matSize));
  // CUDA_ERROR_CHECK(hipHostMalloc((void **) &pinned_imag_image, matSize));
  // memcpy(pinned_real_image, real_image, matSize);
  // memcpy(pinned_imag_image, imag_image, matSize);


  // printf("\n1st row real\n");
  // for (int i = 0; i < SIZE; ++i)
  // {
  //   printf("%f, ", real_image[i]);
  // }
  // printf("\n1st row imag\n");
  // for (int i = 0; i < SIZE; ++i)
  // {
  //   printf("%f, ", imag_image[i]);
  // }

  #define ASYNC_BLOCKS 16

  hipStream_t stream[ASYNC_BLOCKS];
  CUDA_ERROR_CHECK(hipEventRecord(start_bis,filterStream));
  for (int i = 0; i < ASYNC_BLOCKS; ++i)
  {
    hipStreamCreate(&stream[i]);
  }

  CUDA_ERROR_CHECK(hipEventRecord(stop_bis,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop_bis));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&stream_creation,start_bis,stop_bis));

  CUDA_ERROR_CHECK(hipEventRecord(start_bis,filterStream));

  for (int i = 0; i < ASYNC_BLOCKS; ++i)
  {
    CUDA_ERROR_CHECK(hipMemcpyAsync(device_real + i * SIZE*SIZE/ASYNC_BLOCKS, real_image + i * SIZE*SIZE/ASYNC_BLOCKS,matSize/ASYNC_BLOCKS,hipMemcpyHostToDevice, stream[i]));
    CUDA_ERROR_CHECK(hipMemcpyAsync(device_imag + i * SIZE*SIZE/ASYNC_BLOCKS, imag_image + i * SIZE*SIZE/ASYNC_BLOCKS,matSize/ASYNC_BLOCKS,hipMemcpyHostToDevice, stream[i]));
    forwardFFTRow<<<SIZE / ASYNC_BLOCKS, SIZE, 0, stream[i]>>>(device_real + i * SIZE*SIZE/ASYNC_BLOCKS, device_imag + i * SIZE*SIZE/ASYNC_BLOCKS);
  }

  CUDA_ERROR_CHECK(hipDeviceSynchronize());


  // forwardFFTRow<<<SIZE, SIZE, 0, filterStream>>>(device_real, device_imag);

  // CUDA_ERROR_CHECK(hipMemcpy(real_image,device_real,matSize,hipMemcpyDeviceToHost));
  // CUDA_ERROR_CHECK(hipMemcpy(imag_image,device_imag,matSize,hipMemcpyDeviceToHost));

  // printf("\n1st row tranform real\n");
  // for (int i = 0; i < SIZE; ++i)
  // {
  //   printf("%f, ", real_image[i]);
  // }
  // printf("\n1st row tranform imag\n");
  // for (int i = 0; i < SIZE; ++i)
  // {
  //   printf("%f, ", imag_image[i]);
  // }

  CUDA_ERROR_CHECK(hipEventRecord(stop_bis,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop_bis));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&fftr,start_bis,stop_bis));

  CUDA_ERROR_CHECK(hipEventRecord(start_bis,filterStream));

  forwardFFTCol<<<SIZE / 4, SIZE, 0, filterStream>>>(device_real, device_imag);
  CUDA_ERROR_CHECK(hipEventRecord(stop_bis,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop_bis));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&fftc,start_bis,stop_bis));

  CUDA_ERROR_CHECK(hipEventRecord(start_bis,filterStream));
  filter<<<SIZE, SIZE, 0, filterStream>>>(device_real, device_imag);
  CUDA_ERROR_CHECK(hipEventRecord(stop_bis,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop_bis));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&filter_time,start_bis,stop_bis));

  CUDA_ERROR_CHECK(hipEventRecord(start_bis,filterStream));
  inverseFFTCol<<<SIZE / 4, SIZE, 0, filterStream>>>(device_real, device_imag);
  CUDA_ERROR_CHECK(hipEventRecord(stop_bis,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop_bis));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&ifftc,start_bis,stop_bis));

  CUDA_ERROR_CHECK(hipEventRecord(start_bis,filterStream));
  for (int i = 0; i < ASYNC_BLOCKS; ++i)
  {
  }
  for (int i = 0; i < ASYNC_BLOCKS; ++i)
  {
    inverseFFTRow<<<SIZE / ASYNC_BLOCKS, SIZE, 0, stream[i]>>>(device_real + i * SIZE*SIZE/ASYNC_BLOCKS, device_imag + i * SIZE*SIZE/ASYNC_BLOCKS);
    CUDA_ERROR_CHECK(hipMemcpyAsync(real_image + i * SIZE*SIZE/ASYNC_BLOCKS,device_real + i * SIZE*SIZE/ASYNC_BLOCKS,matSize/ASYNC_BLOCKS,hipMemcpyDeviceToHost, stream[i]));
    CUDA_ERROR_CHECK(hipMemcpyAsync(imag_image + i * SIZE*SIZE/ASYNC_BLOCKS,device_imag + i * SIZE*SIZE/ASYNC_BLOCKS,matSize/ASYNC_BLOCKS,hipMemcpyDeviceToHost, stream[i]));
    // CUDA_ERROR_CHECK(hipMemcpyAsync(pinned_real_image + i * SIZE*SIZE/ASYNC_BLOCKS,device_real + i * SIZE*SIZE/ASYNC_BLOCKS,matSize/ASYNC_BLOCKS,hipMemcpyDeviceToHost, stream[i]));
    // CUDA_ERROR_CHECK(hipMemcpyAsync(pinned_imag_image + i * SIZE*SIZE/ASYNC_BLOCKS,device_imag + i * SIZE*SIZE/ASYNC_BLOCKS,matSize/ASYNC_BLOCKS,hipMemcpyDeviceToHost, stream[i]));
  }

  CUDA_ERROR_CHECK(hipDeviceSynchronize());

  // inverseFFTRow<<<SIZE , SIZE, 0, filterStream>>>(device_real, device_imag);
  CUDA_ERROR_CHECK(hipEventRecord(stop_bis,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop_bis));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&ifftr,start_bis,stop_bis));


  //---------------------------------------------------------------- 
  // END ADD KERNEL CALLS
  //----------------------------------------------------------------

  // Finish timimg for the execution 
  CUDA_ERROR_CHECK(hipEventRecord(stop_bis,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop_bis));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&execution,start,stop_bis));

  // Start timing for the transfer up
  CUDA_ERROR_CHECK(hipEventRecord(start,filterStream));

  // Here is where we copy matrices back from the device 
  // CUDA_ERROR_CHECK(hipMemcpy(real_image,device_real,matSize,hipMemcpyDeviceToHost));
  // CUDA_ERROR_CHECK(hipMemcpy(imag_image,device_imag,matSize,hipMemcpyDeviceToHost));

  // Finish timing for transfer up
  CUDA_ERROR_CHECK(hipEventRecord(stop,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&transferUp,start,stop));

  // Synchronize the stream
  CUDA_ERROR_CHECK(hipStreamSynchronize(filterStream));
  // Destroy the stream
  CUDA_ERROR_CHECK(hipStreamDestroy(filterStream));
  for (int i = 0; i < ASYNC_BLOCKS; ++i)
    hipStreamDestroy(stream[i]);
  // Destroy the events
  CUDA_ERROR_CHECK(hipEventDestroy(start));
  CUDA_ERROR_CHECK(hipEventDestroy(stop));

  // Free the memory
  CUDA_ERROR_CHECK(hipFree(device_real));
  CUDA_ERROR_CHECK(hipFree(device_imag));

  // Dump some usage statistics
  printf("CUDA IMPLEMENTATION STATISTICS:\n");
  printf("  Host to Device Transfer Time: %f ms\n", transferDown);
  printf("  Kernel(s) Execution Time: %f ms\n", execution);
  printf("  Device to Host Transfer Time: %f ms\n", transferUp);
  float totalTime = transferDown + execution + transferUp;
  printf("  Total CUDA Execution Time: %f ms\n\n", totalTime);

  printf("  Stream Creation Time: %f ms\n\n", stream_creation);
  printf("  Row DFT Time: %f ms\n\n", fftr);
  printf("  Col DFT Time: %f ms\n\n", fftc);
  printf("  Filter Time: %f ms\n\n", filter_time);
  printf("  Col IDFT Time: %f ms\n\n", ifftc);
  printf("  Row IDFT Time: %f ms\n\n", ifftr);
  // Return the total time to transfer and execute
  return totalTime;
}

